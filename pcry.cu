#include "hip/hip_runtime.h"
#include "pcry.cuh"

__global__ void pcryCalculate_ACC(
                  float* dustPosX, float* dustPosY, float* dustPosZ,
			  	  float* dustVelX, float* dustVelY, float* dustVelZ,
			  	  float* dustAccX, float* dustAccY, float* dustAccZ,
                  float* dustRadius,
                  float* dustMass,
                  float* dustCharge,
                  float* wakeCharge,
                  float* wakeLength,
                  float* wakeDistanceZ,
                  float* wakeDistanceR,
			  	  int* wakeID, 
			  	  float DUST_RADIUS_MEAN,
				  float COULOMB, 
				  float DEBYE, 
				  float CUTOFF_M, 
				  float CELL_RADIUS, 
				  float CELL_CHARGE, 
				  float CELL_HEIGHT, 
				  float SHEATH_HEIGHT, 
				  float WAKE_CHARGE_PERCENT,
				  float GRAVITY, 
				  float GAS_TEMP,
                  float GAS_PRESSURE,
                  float TIME_STEP,
                  double TIME,
				  int 	NUM_PARTICLES){
			  
	// VARIABLE DICTIONARY----------------------------------------------------------------------------------------------
	float 	acc;            // Temporarily stores acceleration
    float   epsilon;        // Softening factor
    float   Ez;             // Electric field by the lower electrode

    // ITH PARTICLE
    float   x1;             // Stores the x position of the ith particle
    float   y1;             // Stores the y position of the ith particle
    float   z1;             // Stores the z position of the ith particle
    float   ax1;            // x component of the accelaration of the ith particle
    float   ay1;            // y component of the acceleration of the ith particle
    float   az1;            // z component of the acceleration of the ith particle
    float   mass1;          // mass of the ith particle
    float   charge1;        // charge of the ith particle
    float   radius1;        // radius of the ith particle

    // JTH PARTICLE         !! This part should actually not be here, what wee need to replace is the variables below
    float   x2;             // Stores the x position of the jth particle
    float   y2;             // Stores the y position of the jth particle
    float   z2;             // z position of the jth particle

    // OTHER VARIABLES
    float   dx;             // Distance between the particles in the x direction
    float   dy;             // Distance between the particles in the y direction
    float   dz;             // Distance between the particles in the z direction
    float   r_squared;      // Norm squared
    float   r;              // Eucledian distance between the particles
    float   r_soft;         // Eucledian distance with softening factor
    float   r_min;
    float   z_min;
    float   yourId;
    float   nn_id;

	float 	accX_i, accY_i, accZ_i; 
	float 	posX_i, posY_i, posZ_i;
	float 	charge_i, mass_i;

    float   SIGMA;
    float   BETA;

    // STATE FOR GENERATING RANDOM NUMBER
    hiprandState_t state;


	// VARIABLES TO BE ALLOCATED IN SHARED MEMORY
	__shared__ float posX_j[BLOCK], posY_j[BLOCK], posZ_j[BLOCK];
	__shared__ float charge_j[BLOCK], wakeCharge_j[BLOCK], wakeLength_j[BLOCK];

    //------------------------------------------------------------------------------------------------------------------
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i < NUM_PARTICLES){              // Making sure we are not out working past the number of particles
        epsilon  = 1e-6;

		// Save positions
        x1  = dustPosX[i];
        y1  = dustPosY[i];
        z1  = dustPosZ[i];
        
		posX_i 	 = dustPosX[i];
		posY_i 	 = dustPosY[i];
		posZ_i 	 = dustPosZ[i];
		
		// Load other attributes
		charge_i = dustCharge[i];
		mass_i	 = dustMass[i];

        mass1   = dustMass[i];
        charge1 = dustCharge[i];
        radius1 = dustRadius[i];
		
		// Initialize forces
		accX_i 	 = 0.0f;
		accY_i 	 = 0.0f;
		accZ_i 	 = 0.0f;
		
		// Other variables
		r_min 	 = CUTOFF_M * DEBYE;
		z_min 	 = 100000.0;
		nn_id 	 = -1;

		wakeDistanceR[i] = 10000.0f;
		wakeDistanceZ[i] = 10000.0f;
		wakeID[i]	 = -1;
		
		// CALCULATING INTERPARTICLE FORCES-----------------------------------------------------------------------------
		for(int j = 0; j < gridDim.x; j++){
			// Save into shared memory
			posX_j[threadIdx.x] 	= dustPosX[threadIdx.x + blockDim.x*j];
			posY_j[threadIdx.x] 	= dustPosY[threadIdx.x + blockDim.x*j];
			posZ_j[threadIdx.x] 	= dustPosZ[threadIdx.x + blockDim.x*j];
			
			charge_j[threadIdx.x] 	= dustCharge[threadIdx.x + blockDim.x*j];
			wakeCharge_j[threadIdx.x] = wakeCharge[threadIdx.x + blockDim.x*j];
			wakeLength_j[threadIdx.x] = wakeLength[threadIdx.x + blockDim.x*j];
			
			__syncthreads();
			
			#pragma unroll 32
            for(int yourSharedId = 0; yourSharedId < blockDim.x; yourSharedId++){
                yourId = yourSharedId + blockDim.x*j;
		    		
                if(i != yourId && yourId < NUM_PARTICLES){
                    dx 		= posX_j[yourSharedId] - posX_i;
                    dy		= posY_j[yourSharedId] - posY_i;
                    dz 		= posZ_j[yourSharedId] - posZ_i;

                    // Norm squared with smoothing factor
                    r_squared  	= dx*dx + dy*dy + dz*dz;
                    r_soft      = sqrt(r_squared + (epsilon * epsilon));
                    r		= sqrt(r_squared);
                    
                    // DUST-DUST YUKAWA FORCE
                    acc  = -COULOMB*charge_j[yourSharedId]*charge_i*(1.0f+r/DEBYE)*exp(-r/DEBYE)/(r_soft*r_soft);
                    acc /= (mass_i);

                    accX_i	+= acc * (dx/r_soft);
                    accY_i 	+= acc * (dy/r_soft);
                    accZ_i 	+= acc * (dz/r_soft);

                    // Finding the nearest neighbor below the current dust grain and within
                    // the specified distance (6 * DEBYE)
                    // We will use this to set the ionWake of the two dusts in question.
                    // This will be done in the move function to remove any race conditions.
                    if(dz < 0.0f){ // If dz is negative you are below me.
                        if(r < r_min){
                            r_min	= r;
                            z_min	= dz;
                            nn_id   = yourId;  // This needs to be the real Id not what is in shared memory.
                        }
                    }
                    


                    // DUST-ION YUKAWA FORCE
                    dz = (posZ_j[yourSharedId] - wakeLength_j[yourSharedId]) - posZ_i;

                    r_squared  	= dx*dx + dy*dy + dz*dz;
                    r_soft	= sqrt(r_squared + (epsilon * epsilon));
                    r  	   	= sqrt(r_squared);

                    acc 	= (COULOMB*charge_j[yourSharedId]*wakeCharge_j[yourSharedId]*charge_i)/(r_soft*r_soft);
                    acc		*=(1.0f + r/DEBYE)*exp(-r/DEBYE)/mass_i;

                    accX_i 	+= acc * (dx/r_soft);
                    accY_i 	+= acc * (dy/r_soft);
                    accZ_i 	+= acc * (dz/r_soft);
				}
			}
		}
		
		wakeDistanceR[i]    		= r_min;	// Saving minimum total distance
		wakeDistanceZ[i]			= z_min;	// Saving minimum y distance
		wakeID[i]			 	    = nn_id; 	// Saving the nearest neighbor's ID

        // CALCULATING EXTERNAL FORCES----------------------------------------------------------------------------------
		Ez = -8083 + 553373*z1 + 2.0e8*(z1*z1) - 3.017e10*pow(z1,3) + 1.471e12*pow(z1,4) - 2.306e13*pow(z1,5);
		accZ_i += charge1 * Ez / mass1;
		
		// RADIAL CONFINEMENT FORCE
		r  	= sqrt(x1*x1+y1*y1);
		if(r != 0){
		    acc = charge_i*CELL_CHARGE*pow(r/CELL_RADIUS,12)/mass_i;
		    accX_i += acc * (posX_i/r);
		    accY_i += acc * (posY_i/r);
		}

		// GRAVITATIONAL FORCE
		accZ_i += -GRAVITY;
		
		// DRAG FORCE
        BETA = 1.44* 4.0 /3.0 * (radius1*radius1) * GAS_PRESSURE / mass1 * sqrt(8.0 * PI * ION_MASS/BOLTZMANN/GAS_TEMP);

		accX_i += -BETA * dustVelX[i];
		accY_i += -BETA * dustVelY[i];
		accZ_i += -BETA * dustVelZ[i];

        // BROWNIAN MOTION
        hiprand_init((time_t)(TIME+i),0,0,&state);
        SIGMA = sqrt(2.0* BETA * BOLTZMANN * GAS_TEMP/mass1/TIME_STEP);

        accX_i += SIGMA * hiprand_normal(&state);
        accY_i += SIGMA * hiprand_normal(&state);
        accZ_i += SIGMA * hiprand_normal(&state);

        // LOAD FORCES--------------------------------------------------------------------------------------------------
        // If the dust grain gets too close or passes through the floor. I put it at the top of the sheath, set its
        // force to zero and set its mass, charge and diameter to the base (maybe it was too heavy).
		if(DUST_RADIUS_MEAN < posZ_i){
			dustAccX[i] = accX_i;
			dustAccY[i] = accY_i;
			dustAccZ[i] = accZ_i;
		} else{
			dustPosZ[i] = SHEATH_HEIGHT;

			dustVelX[i] 	= 0.0;
			dustVelY[i] 	= 0.0;
			dustVelZ[i] 	= 0.0;
			
			dustAccX[i]     = 0.0;
			dustAccY[i]     = 0.0;
			dustAccZ[i]     = 0.0;
		}
	}
}


__global__ void pcryCalculate_POS(
                  float* dustPosX, float* dustPosY, float* dustPosZ,
				  float* dustVelX, float* dustVelY, float* dustVelZ,
				  float* dustAccX, float* dustAccY, float* dustAccZ,
				  float* dustRadius,
				  float* dustMass,
                  float* dustCharge,
                  float* wakeCharge,
				  float* wakeLength,
                  float* wakeDistanceZ,
                  float* wakeDistanceR,
				  int* wakeID,
			  	  float DUST_CHARGE_DENSITY_MEAN,
			  	  float ELECTRON_CHARGE,
				  float CUTOFF_M,  
				  float WAKE_CHARGE_PERCENT, 
				  float WAKE_LENGTH,
				  float DEBYE,
				  float DT,
				  float TIME,
				  int 	NUM_PARTICLES){
	// Moving the system forward in time with leap-frog and randomly adjusting the charge on each dust particle.
	hiprandState state;
	float randomNumber;
	float cutOff, reduction;
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	
	// Note: DustForce.w hold the mass of the dust grain.
	if(i < NUM_PARTICLES){
        /*
		// Updating my ionwake percent charge and length below the dust if its distance to the its nearest dust grain is within debyeLengthMultiplier*debyeLengths.
		// Also adding the percent charge that the upstream ionWake lost to the down stream ionWake.
		// Need to do this before you update the positions or you will get a miss read on dy.
		if(wakeDistanceZ[i] < 0.0){ // It was initialized to 100000.0 and if no dust grain is close enough it will stay -100000.0.
			cutOff = CUTOFF_M*DEBYE;
			// This is a quadratic function that goes from 1 to 0 as the dust-dust distance goes from debyeLengthMultiplier*debyeLength to zero.
			// Using a second order because as the bottom dust ets close to the top dust it will be eating up a ring (second order) of ions that would have added to the ionwake.
			// It will be used to decrease the top dust's ionwake and give this lose to the bottom dust's ionwake.
			reduction = (1.0f - wakeDistanceR[i]*wakeDistanceR[i]/(cutOff*cutOff))*(wakeDistanceZ[i] /
				     wakeDistanceR[i])*(wakeDistanceZ[i]/wakeDistanceR[i]);
				     
			
			wakeCharge[i]		    = WAKE_CHARGE_PERCENT - WAKE_CHARGE_PERCENT*reduction;	// Decreasing the top charge
			wakeCharge[wakeID[i]]	= WAKE_CHARGE_PERCENT + WAKE_CHARGE_PERCENT*reduction;	// Increasing the bottom charge
			
			// This is a linear that goes from 1 to 0 as the dust-dust distance goes from debyeLengthMultiplier*debyeLength to zero.
			// Using a first order because the as the bottom dust moves up linearly it will displace a ring of ions that would have added to the ionwake.
			// It will be used to decrease the top dust's ionwake length below the top dust.
			//reduction = ionWake[id].z/cutOff;
			reduction = (1.0f - wakeDistanceR[i]/(cutOff))*(wakeDistanceZ[i]/wakeDistanceR[i])*(wakeDistanceZ[i]/wakeDistanceR[i]);
			wakeLength[i] = WAKE_LENGTH - WAKE_LENGTH*reduction;
		} else{
			// If for some reason the ionwake didn't get turned back on it is reset here.
			wakeCharge[i] = WAKE_CHARGE_PERCENT;
			wakeLength[i] = WAKE_LENGTH;
		}	*/
		
		if(TIME == 0.0f){
			dustVelX[i] += 0.5f*DT*dustAccX[i];
			dustVelY[i] += 0.5f*DT*dustAccY[i];
			dustVelZ[i] += 0.5f*DT*dustAccZ[i];
		} else {
			dustVelX[i] += DT*dustAccX[i];
			dustVelY[i] += DT*dustAccY[i];
			dustVelZ[i] += DT*dustAccZ[i];
		}

		dustPosX[i] += dustVelX[i]*DT;
		dustPosY[i] += dustVelY[i]*DT;
		dustPosZ[i] += dustVelZ[i]*DT;

        /*
		// Randomly perturbating the dust electron count. 
		// This gets a little involved. I first get a standard normal distributed number (Mean 0 StDev 1).
		// Then I set its StDev to the number of electrons that fluctuate per unit dust diameter for this dust grain size.
		// Then I set the mean to how much above or below the base electron per unit dust size.
		// ie. if it has more than it should it has a higher prob of losing and vice versa if it has less than it should.
		// This is just what I came up with and it could be wrong but below is how I did this.
		// dustPos.w carries the charge and dustVel.w carries the diameter.
		
		// Initailizing the cudarand function.
		hiprand_init(clock64(), i, 0, &state);
		// This gets a random number with mean 0.0 and stDev 1.0;.
		randomNumber = hiprand_normal(&state);
		// This sets the electron fluctuation for this sized dust grain and makes it the stDev.
		randomNumber *= DUST_CHARGE_DENSITY_MEAN*(2 * dustRadius[i]);
		
		// This has a mean of zero which would just create a random walk. I don't think this is what you want.
		// Dust grains with more electrons than they should have should in general loose electrons 
		// and those with less than they should should in general gain more electrons.
		// We will accomplish this by setting the mean to be the oposite of how much above or below 
		// the base amount you are at this time.
		// This works out to be base number - present number
		randomNumber += DUST_CHARGE_DENSITY_MEAN*(2*dustRadius[i]) - dustCharge[i]/ELECTRON_CHARGE;
		
		// Now add/subtract this number of electron to the existing charge.
    		dustCharge[i] += randomNumber*ELECTRON_CHARGE;
	   
	    	// If the amount of charge ends up being negative which probablistically it could, set it to zero
	    	if(dustCharge[i] < 0.0) dustCharge[i] = 0.0;*/
	}				
}








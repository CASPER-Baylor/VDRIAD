#include "hip/hip_runtime.h"
#include "pcry.cuh"

__global__ void pcryCalculate_ACC(
                  float* dustPosX, float* dustPosY, float* dustPosZ,
			  	  float* dustVelX, float* dustVelY, float* dustVelZ,
			  	  float* dustAccX, float* dustAccY, float* dustAccZ,
                  float* dustRadius,
                  float* dustMass,
                  float* dustCharge,
                  float* wakeCharge,
                  float* wakeLength,
                  float* wakeDistanceZ,
                  float* wakeDistanceR,
			  	  int* wakeID, 
			  	  float DUST_RADIUS_MEAN,
				  float COULOMB, 
				  float DEBYE, 
				  float CUTOFF_M, 
				  float CELL_RADIUS, 
				  float CELL_CHARGE, 
				  float CELL_HEIGHT, 
				  float SHEATH_HEIGHT, 
				  float WAKE_CHARGE_PERCENT,
				  float GRAVITY, 
				  float BETA,
				  int 	NUM_PARTICLES){
			  
	// VARIABLE DICTIONARY----------------------------------------------------------------------------------------------
	float 	acc;            // Temporarily stores acceleration
    float   epsilon;        // Softening factor
    float   Ez;             // Electric field by the lower electrode

    // ITH PARTICLE
    float   x1;             // Stores the x position of the ith particle
    float   y1;             // Stores the y position of the ith particle
    float   z1;             // Stores the z position of the ith particle
    float   ax1;            // x component of the accelaration of the ith particle
    float   ay1;            // y component of the acceleration of the ith particle
    float   az1;            // z component of the acceleration of the ith particle
    float   mass1;          // mass of the ith particle
    float   charge1;        // charge of the ith particle

    // JTH PARTICLE
    float   x2;             // Stores the x position of the jth particle
    float   y2;             // Stores the y position of the jth particle
    float   z2;             // z position of the jth particle
    float   ax2;            // x component of the acceleration of the jth particle
    float   ay2;            // y component of the acceleration of the jth particle
    float   az2;            // z component of the acceleration of the jth particle

    // OTHER VARIABLES
    float   dx;             // Distance between the particles in the x direction
    float   dy;             // Distance between the particles in the y direction
    float   dz;             // Distance between the particles in the z direction
    float   r_squared;      // Norm squared
    float   r;              // Eucledian distance between the particles
    float   r_soft;         // Eucledian distance with softening factor
    float   r_min;
    float   z_min;
    float   yourId;
    float   nn_id;


    float   zi,zi2;
	float 	accX_i, accY_i, accZ_i; 
	float 	posX_i, posY_i, posZ_i;
	float 	charge_i, mass_i;

	// VARIABLES TO BE ALLOCATED IN SHARED MEMORY
	__shared__ float posX_j[BLOCK], posY_j[BLOCK], posZ_j[BLOCK];
	__shared__ float charge_j[BLOCK], wakeCharge_j[BLOCK], wakeLength_j[BLOCK];

    //------------------------------------------------------------------------------------------------------------------
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	if(i < NUM_PARTICLES){              // Making sure we are not out working past the number of particles.
        epsilon  = 2 * dustRadius[i];

		// Save positions
		posX_i 	 = dustPosX[i];
		posY_i 	 = dustPosY[i];
		posZ_i 	 = dustPosZ[i];
		
		// Load other attributes
		charge_i = dustCharge[i];
		mass_i	 = dustMass[i];
		
		// Initialize forces
		accX_i 	 = 0.0f;
		accY_i 	 = 0.0f;
		accZ_i 	 = 0.0f;
		
		// Other variables
		r_min 	 = CUTOFF_M * DEBYE;
		z_min 	 = 100000.0;
		nn_id 	 = -1;

		wakeDistanceR[i] = 10000.0f;
		wakeDistanceZ[i] = 10000.0f;
		wakeID[i]	 = -1;
		
		// CALCULATING INTERPARTICLE FORCES-----------------------------------------------------------------------------
		for(int j = 0; j < gridDim.x; j++){
			// Save into shared memory
			posX_j[threadIdx.x] 	= dustPosX[threadIdx.x + blockDim.x*j];
			posY_j[threadIdx.x] 	= dustPosY[threadIdx.x + blockDim.x*j];
			posZ_j[threadIdx.x] 	= dustPosZ[threadIdx.x + blockDim.x*j];
			
			charge_j[threadIdx.x] 	= dustCharge[threadIdx.x + blockDim.x*j];
			wakeCharge_j[threadIdx.x] = wakeCharge[threadIdx.x + blockDim.x*j];
			wakeLength_j[threadIdx.x] = wakeLength[threadIdx.x + blockDim.x*j];
			
			__syncthreads();
			
			#pragma unroll 32
            for(int yourSharedId = 0; yourSharedId < blockDim.x; yourSharedId++){
                yourId = yourSharedId + blockDim.x*j;
		    		
                if(i != yourId && yourId < NUM_PARTICLES){
                    dx 		= posX_j[yourSharedId] - posX_i;
                    dy		= posY_j[yourSharedId] - posY_i;
                    dz 		= posZ_j[yourSharedId] - posZ_i;

                    // Norm squared with smoothing factor
                    r_squared  	= dx*dx + dy*dy + dz*dz;
                    r_soft      = sqrt(r_squared + (epsilon * epsilon));
                    r		    = sqrt(r_squared);
                    
                    // DUST-DUST YUKAWA FORCE
                    acc  = -COULOMB*charge_j[yourSharedId]*charge_i*(1.0f + r/DEBYE)*exp(-r/DEBYE)/r_soft;
                    acc /= (mass_i);


                    accX_i		+= acc * (dx/r);
                    accY_i 		+= acc * (dy/r);
                    accZ_i 		+= acc * (dz/r);

                    // Finding the nearest neighbor below the current dust grain and within
                    // the specified distance (6 * DEBYE)
                    // We will use this to set the ionWake of the two dusts in question.
                    // This will be done in the move function to remove any race conditions.
                    if(dz < 0.0f){ // If dz is negative you are below me.
                        if(r < r_min){
                            r_min	= r;
                            z_min	= dz;
                            nn_id   = yourId;  // This needs to be the real Id not what is in shared memory.
                        }
                    }
                    


                    // DUST-ION YUKAWA FORCE
                    dz = (posZ_j[yourSharedId] - wakeLength_j[yourSharedId]) - posZ_i;

                    r_squared  	= dx*dx + dy*dy + dz*dz + 1.0e-6;
                    r  	   	= sqrt(r_squared);

                    acc 	= (COULOMB*charge_j[yourSharedId]*wakeCharge_j[yourSharedId]*charge_i)/r_squared;
                    acc		*=(1.0f + r/DEBYE)*exp(-r/DEBYE)/mass_i;

                    accX_i 		+= acc * (dx/r);
                    accY_i 		+= acc * (dy/r);
                    accZ_i 		+= acc * (dz/r);
				}
			}
		}
		
		wakeDistanceR[i]    		= r_min;	// Saving minimum total distance
		wakeDistanceZ[i]			= z_min;	// Saving minimum y distance
		wakeID[i]			 	    = nn_id; 	// Saving the nearest neighbor's ID

        // CALCULATING EXTERNAL FORCES----------------------------------------------------------------------------------
        zi = posZ_i;
        zi2 = zi * zi;

		Ez = -8083 + 553373*zi + 2.0e8*zi2 -3.017e10*zi*zi2 + 1.471e12*zi2*zi2 - 2.306e13*zi2*zi2*zi;
		accZ_i += charge_i * Ez / mass_i;
		
		// Calculating radial confinement force
		r  	= sqrt(posX_i*posX_i + posY_i*posY_i);
		if(r != 0){
		    acc = charge_i*CELL_CHARGE*pow(r/CELL_RADIUS,12)/mass_i;
		    accX_i += acc * (posX_i/r);
		    accY_i += acc * (posY_i/r);
		}

		
		// Calculate acceleration due to gravity
		accZ_i += -GRAVITY;
		
		// Calculating drag force
		accX_i += -BETA * dustVelX[i];
		accY_i += -BETA * dustVelY[i];
		accZ_i += -BETA * dustVelZ[i];
		
		if(DUST_RADIUS_MEAN < posZ_i){
			dustAccX[i] = accX_i;
			dustAccY[i] = accY_i;
			dustAccZ[i] = accZ_i;
		} else{ // If the dust grain gets too close or passes through the floor. I put it at the top of the sheath, set its force to zero and set its mass, charge and diameter to the base (maybe it was too heavy).
			dustPosZ[i] = SHEATH_HEIGHT;

			dustVelX[i] 	= 0.0;
			dustVelY[i] 	= 0.0;
			dustVelZ[i] 	= 0.0;
			
			dustAccX[i]     = 0.0;
			dustAccY[i]     = 0.0;
			dustAccZ[i]     = 0.0;
		}
	}
}


__global__ void pcryCalculate_POS(
                  float* dustPosX, float* dustPosY, float* dustPosZ,
				  float* dustVelX, float* dustVelY, float* dustVelZ,
				  float* dustAccX, float* dustAccY, float* dustAccZ,
				  float* dustRadius,
				  float* dustMass,
                  float* dustCharge,
                  float* wakeCharge,
				  float* wakeLength,
                  float* wakeDistanceZ,
                  float* wakeDistanceR,
				  int* wakeID,
			  	  float DUST_CHARGE_DENSITY_MEAN,
			  	  float ELECTRON_CHARGE,
				  float CUTOFF_M,  
				  float WAKE_CHARGE_PERCENT, 
				  float WAKE_LENGTH,
				  float DEBYE,
				  float DT,
				  float TIME,
				  int 	NUM_PARTICLES){
	// Moving the system forward in time with leap-frog and randomly adjusting the charge on each dust particle.
	hiprandState state;
	float randomNumber;
	float cutOff, reduction;
	int i = threadIdx.x + blockDim.x*blockIdx.x;
	
	// Note: DustForce.w hold the mass of the dust grain.
	if(i < NUM_PARTICLES){
        /*
		// Updating my ionwake percent charge and length below the dust if its distance to the its nearest dust grain is within debyeLengthMultiplier*debyeLengths.
		// Also adding the percent charge that the upstream ionWake lost to the down stream ionWake.
		// Need to do this before you update the positions or you will get a miss read on dy.
		if(wakeDistanceZ[i] < 0.0){ // It was initialized to 100000.0 and if no dust grain is close enough it will stay -100000.0.
			cutOff = CUTOFF_M*DEBYE;
			// This is a quadratic function that goes from 1 to 0 as the dust-dust distance goes from debyeLengthMultiplier*debyeLength to zero.
			// Using a second order because as the bottom dust ets close to the top dust it will be eating up a ring (second order) of ions that would have added to the ionwake.
			// It will be used to decrease the top dust's ionwake and give this lose to the bottom dust's ionwake.
			reduction = (1.0f - wakeDistanceR[i]*wakeDistanceR[i]/(cutOff*cutOff))*(wakeDistanceZ[i] /
				     wakeDistanceR[i])*(wakeDistanceZ[i]/wakeDistanceR[i]);
				     
			
			wakeCharge[i]		    = WAKE_CHARGE_PERCENT - WAKE_CHARGE_PERCENT*reduction;	// Decreasing the top charge
			wakeCharge[wakeID[i]]	= WAKE_CHARGE_PERCENT + WAKE_CHARGE_PERCENT*reduction;	// Increasing the bottom charge
			
			// This is a linear that goes from 1 to 0 as the dust-dust distance goes from debyeLengthMultiplier*debyeLength to zero.
			// Using a first order because the as the bottom dust moves up linearly it will displace a ring of ions that would have added to the ionwake.
			// It will be used to decrease the top dust's ionwake length below the top dust.
			//reduction = ionWake[id].z/cutOff;
			reduction = (1.0f - wakeDistanceR[i]/(cutOff))*(wakeDistanceZ[i]/wakeDistanceR[i])*(wakeDistanceZ[i]/wakeDistanceR[i]);
			wakeLength[i] = WAKE_LENGTH - WAKE_LENGTH*reduction;
		} else{
			// If for some reason the ionwake didn't get turned back on it is reset here.
			wakeCharge[i] = WAKE_CHARGE_PERCENT;
			wakeLength[i] = WAKE_LENGTH;
		}	*/
		
		if(TIME == 0.0f){
			dustVelX[i] += 0.5f*DT*dustAccX[i];
			dustVelY[i] += 0.5f*DT*dustAccY[i];
			dustVelZ[i] += 0.5f*DT*dustAccZ[i];
		} else {
			dustVelX[i] += DT*dustAccX[i];
			dustVelY[i] += DT*dustAccY[i];
			dustVelZ[i] += DT*dustAccZ[i];
		}

		dustPosX[i] += dustVelX[i]*DT;
		dustPosY[i] += dustVelY[i]*DT;
		dustPosZ[i] += dustVelZ[i]*DT;

        /*
		// Randomly perturbating the dust electron count. 
		// This gets a little involved. I first get a standard normal distributed number (Mean 0 StDev 1).
		// Then I set its StDev to the number of electrons that fluctuate per unit dust diameter for this dust grain size.
		// Then I set the mean to how much above or below the base electron per unit dust size.
		// ie. if it has more than it should it has a higher prob of losing and vice versa if it has less than it should.
		// This is just what I came up with and it could be wrong but below is how I did this.
		// dustPos.w carries the charge and dustVel.w carries the diameter.
		
		// Initailizing the cudarand function.
		hiprand_init(clock64(), i, 0, &state);
		// This gets a random number with mean 0.0 and stDev 1.0;.
		randomNumber = hiprand_normal(&state);
		// This sets the electron fluctuation for this sized dust grain and makes it the stDev.
		randomNumber *= DUST_CHARGE_DENSITY_MEAN*(2 * dustRadius[i]);
		
		// This has a mean of zero which would just create a random walk. I don't think this is what you want.
		// Dust grains with more electrons than they should have should in general loose electrons 
		// and those with less than they should should in general gain more electrons.
		// We will accomplish this by setting the mean to be the oposite of how much above or below 
		// the base amount you are at this time.
		// This works out to be base number - present number
		randomNumber += DUST_CHARGE_DENSITY_MEAN*(2*dustRadius[i]) - dustCharge[i]/ELECTRON_CHARGE;
		
		// Now add/subtract this number of electron to the existing charge.
    		dustCharge[i] += randomNumber*ELECTRON_CHARGE;
	   
	    	// If the amount of charge ends up being negative which probablistically it could, set it to zero
	    	if(dustCharge[i] < 0.0) dustCharge[i] = 0.0;*/
	}				
}







